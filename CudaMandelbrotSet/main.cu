#include "hip/hip_runtime.h"
﻿#include <Windows.h>
#include <SDL.h>
#include <omp.h>

#include "hip/hip_runtime.h"
#include ""

#define SCREEN_WIDTH 1024
#define SCREEN_HEIGHT 768

SDL_Window* window = nullptr;
SDL_Renderer* renderer = nullptr;


// Инициализирует окно и рендерер
bool initWindow()
{
    bool ok = true;

    if (SDL_Init(SDL_INIT_VIDEO) != 0)
    {
        ok = false;
    }

    window = SDL_CreateWindow("CudaMandelbrot", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH, SCREEN_HEIGHT, SDL_WINDOW_SHOWN);
    if (!window)
    {
        ok = false;
    }

    renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED);
    if (!renderer)
    {
        ok = false;
    }

    return ok;
}

// Возвращает цвет в множестве Мандельброта
__device__ Uint8 getMandelbrotColor(double x, double y)
{
    double z1 = 0, z2 = 0;
    #pragma unroll 4
    for (Uint8 step = 0; step < 100; ++step) 
    {
        double new_z1 = z1 * z1 - z2 * z2 + x;
        double new_z2 = 2 * z1 * z2 + y;
        if ((new_z1 * new_z1 + new_z2 * new_z2) > 4)
            return step * 2.5;
        z1 = new_z1;
        z2 = new_z2;
    }
    return 255;
}

__global__ void computeMandelbrot(Uint8* colors, double min_x, double min_y, const double x_diff, const double y_diff)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < SCREEN_WIDTH && j < SCREEN_HEIGHT)
    {
        double x = min_x + i * x_diff;
        double y = min_y + j * y_diff;
        colors[j * SCREEN_WIDTH + i] = getMandelbrotColor(x, y);
    }
}

// Генерирует множество Мандельброта на экране
void renderMandelbrotSet(double min_x, double max_x, double min_y, double max_y)
{
    const double x_diff = (max_x - min_x) / SCREEN_WIDTH;
    const double y_diff = (max_y - min_y) / SCREEN_HEIGHT;

    Uint8* colors;
    hipMallocManaged(&colors, SCREEN_WIDTH * SCREEN_HEIGHT * sizeof(Uint8));

    dim3 blockSize(16, 16); 
    dim3 gridSize((SCREEN_WIDTH + 15) / 16, (SCREEN_HEIGHT + 15) / 16); 
    computeMandelbrot <<<gridSize, blockSize>>> (colors, min_x, min_y, x_diff, y_diff);
    hipDeviceSynchronize();

    // заполнение пикселей экрана вычисленными цветами
    for (int i = 0; i < SCREEN_HEIGHT; i++) {
        for (int j = 0; j < SCREEN_WIDTH; j++) {
            Uint8 color = colors[i * SCREEN_WIDTH + j];
            SDL_SetRenderDrawColor(renderer, (color * 5) % 256, (color * 7) % 256, (color * 11) % 256, 255);
            SDL_RenderDrawPoint(renderer, j, i);
        }
    }

    SDL_RenderPresent(renderer);
    hipFree(colors);
}

int main(int argc, char* argv[])
{
    ::ShowWindow(::GetConsoleWindow(), SW_HIDE);
    SDL_Init(SDL_INIT_EVERYTHING);
    initWindow();

    // отрезки на которых генерируется множество Мандельброта и шаг передвижения/увеличения
    double curent_min_x = -1, curent_max_x = 1, curent_min_y = -1, curent_max_y = 1;
    double curent_move_step = (curent_max_x - curent_min_x) / 10;

    renderMandelbrotSet(curent_min_x, curent_max_x, curent_min_y, curent_max_y);

    SDL_Event window_event;
    while (true)
    {
        if (SDL_PollEvent(&window_event))
        {
            bool rerender_needed = false;

            // приближение/отдаление
            if (SDL_MOUSEWHEEL == window_event.type)
            {
                if (window_event.wheel.y == -1)
                {
                    curent_min_x -= curent_move_step;
                    curent_max_x += curent_move_step;
                    curent_min_y -= curent_move_step;
                    curent_max_y += curent_move_step;
                }
                else if (window_event.wheel.y == 1)
                {
                    curent_min_x += curent_move_step;
                    curent_max_x -= curent_move_step;
                    curent_min_y += curent_move_step;
                    curent_max_y -= curent_move_step;
                }
                rerender_needed = true;
                curent_move_step = (curent_max_x - curent_min_x) / 10;
            }

            // передвижение на стрелочки
            if (SDL_KEYDOWN == window_event.type)
            {
                if (window_event.key.keysym.sym == SDLK_UP) {
                    curent_min_y -= curent_move_step;
                    curent_max_y -= curent_move_step;
                }
                else if (window_event.key.keysym.sym == SDLK_DOWN) {
                    curent_min_y += curent_move_step;
                    curent_max_y += curent_move_step;
                }
                else if (window_event.key.keysym.sym == SDLK_LEFT) {
                    curent_min_x -= curent_move_step;
                    curent_max_x -= curent_move_step;
                }
                else if (window_event.key.keysym.sym == SDLK_RIGHT) {
                    curent_min_x += curent_move_step;
                    curent_max_x += curent_move_step;
                }
                rerender_needed = true;
                curent_move_step = (curent_max_x - curent_min_x) / 10;
            }

            if (rerender_needed)
                renderMandelbrotSet(curent_min_x, curent_max_x, curent_min_y, curent_max_y);

            if (SDL_QUIT == window_event.type)
                break;
        }
    }

    // Очистка ресурсов
    SDL_DestroyWindow(window);
    SDL_DestroyRenderer(renderer);
    SDL_Quit();
    return EXIT_SUCCESS;
}